#include "hip/hip_runtime.h"
//srun -c 1 -G 1 --pty bash
#include <stdio.h>
#include <memory>
#include <random>
#include <math.h>
#include <vector>
#include <limits>

#include "Weaver.h"
#include "lodepng.h"

bool loadImage(float** data, uint* width, uint* height, const char* fileName) {
	unsigned char* rawImgData;
	unsigned int error = lodepng_decode32_file(&rawImgData, width, height, fileName);
    if(error) {
        printf("decoder error %u: %s\n", error, lodepng_error_text(error));
        return true;
    }

	*data = (float*)malloc((*width) * (*height) * sizeof(float) * 4);

	for (size_t y = 0; y < *height; y++)
	{
		for (size_t x = 0; x < *width; x++)
		{
			int index = ((y * (*width)) + x);
			(*data)[index] = 1.5f * (rawImgData[(index * 4)] + rawImgData[(index * 4) + 1] + rawImgData[(index * 4) + 2]) / 3.0f / 255.0f;
		}
		
	}

	free(rawImgData);
	return false;
}

float* rescale(float* originalImage, uint width, uint height, uint desiredDim) {
	float* newImage = (float*)malloc(desiredDim * desiredDim * sizeof(float));
	int originalSize = min(width, height);
	float scaling = desiredDim / (float)originalSize;

	for (int y = 0; y < desiredDim; y++)
	{
		for (int x = 0; x < desiredDim; x++)
		{
			int ox = (int)(x / scaling);
			int oy = (int)(y / scaling);
			newImage[(y * desiredDim) + x] = originalImage[(oy * width) + ox];
		}
	}

	return newImage;
}

Point* getCircumfrancePoints(int n) {
	const float RADIUS = 0.48f;

	Point* points = (Point*)malloc(sizeof(Point) * n);
	for(int i = 0; i < n; i++) {
		float r = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
		float index = i + ((r - 0.5) * 0.4);
		points[i].x = 0.5 + (RADIUS * cosf32( (index / (float)n) * 6.28318530718f));
		points[i].y = 0.5 + (RADIUS * sinf32( (index / (float)n) * 6.28318530718f));
	}

	return points;
}

int main(int argc, char *argv[]) {
	float* data;
	uint width, height;
	const char* infilename = argc > 1 ? argv[1] : "input.png";
	const char* outfilename = argc > 2 ? argv[2] : "output.png";

	uint pointCount = 102;
	float blurRadius = 0.001f;
	uint iterations = 5000;
	float lineThickness = 0.0005f;
	uint resolution = 512;


	for (size_t i = 3; i < argc; i++)
	{
		std::cout << argv[i] << std::endl;
		if(strcmp(argv[i], "-p") == 0) {
			pointCount = atoi(argv[++i]);
		}

		if(strcmp(argv[i], "-b") == 0) {
			blurRadius = (float)atof(argv[++i]);
		}

		if(strcmp(argv[i], "-i") == 0) {
			iterations = atoi(argv[++i]);
		}

		if(strcmp(argv[i], "-l") == 0) {
			lineThickness = (float)atof(argv[++i]);
		}

		if(strcmp(argv[i], "-r") == 0) {
			resolution = atoi(argv[++i]);
		}
	}
	

	if(loadImage(&data, &width, &height, infilename)) {
		return;
	}

	data = rescale(data, width, height, resolution);

	Point* points = getCircumfrancePoints(pointCount);
	
	Weaver weaver = Weaver(data, points, resolution, pointCount, lineThickness, blurRadius);
	float prevLoss= std::numeric_limits<float>::max();
	int times = 0;
	const int MAX_FAIL_TIMES = 5;
	for (size_t i = 0; i < iterations; i++)
	{
		float loss = weaver.weaveIteration();
		if (prevLoss - loss < 0.01f) {
			if (++times >= MAX_FAIL_TIMES)
				break;
		} else {
			times = 0;
		}
		std::cout << i << ": " << loss << std::endl;
		prevLoss = loss;
	}	
	weaver.saveCurrentImage(outfilename);

	return 0;
}