#include "hip/hip_runtime.h"
#include "WeaveKernel.h"
#include "math.h"
#include "hip/hip_runtime.h"
#include ""

__device__ bool insideLine(float px, float py, float ax, float ay, float bx, float by, float lineThickness) {
    float vAPx = px - ax;
    float vAPy = py - ay;
    float vABx = bx - ax;
    float vABy = by - ay;

    float sqDist = (vABx * vABx) + (vABy * vABy);
    float abaProd = (vABx * vAPx) + (vABy * vAPy);
    float amount = abaProd / sqDist;

    amount = fminf(fmaxf(amount, 0.0f), 1.0f);

    float nx = (amount * (bx - ax)) + ax;
    float ny = (amount * (by - ay)) + ay;

    float dist = sqrtf(((py - ny) * (py - ny)) + ((px - nx) * (px - nx)));
    return dist < lineThickness / 2.0f;
}

const int SAMPLE_GRID_SIZE = 8;
// __device__ float d_aa4x[4] = {0.1f, 0.6f, 0.9f, 0.4f};
// __device__ float d_aa4y[4] = {0.6f, 0.9f, 0.4f, 0.41};

__global__ void dev_drawLine(float* d_weaveBlock, 
    float* d_currentImage,
    Point* d_points,
    int currentPoint,
    int pointCount,
    int resolution,
    float lineThickness) {
    
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int z = (blockIdx.z * blockDim.z) + threadIdx.z;

    float px = x / (float) resolution;
    float py = y / (float) resolution;

    // Ensure thread is within bound
    if(z < pointCount && x < resolution && y < resolution) {

        if(((px - 0.5f) * (px - 0.5f)) + ((py - 0.5f) * (py - 0.5f)) >= 0.25f) {
            return;
        }

        // Draw Line
        float ax = d_points[currentPoint].x;
        float ay = d_points[currentPoint].y;
        float bx = d_points[z].x;
        float by = d_points[z].y;

        float val = 0.0f;
        float max = 0.0f;

        // Perform antialiasing
        for (size_t xa = 0; xa < SAMPLE_GRID_SIZE; xa++)
        {
            for (size_t ya = 0; ya < SAMPLE_GRID_SIZE; ya++)
            {
                if(((ya * SAMPLE_GRID_SIZE) + xa) % 5 == 0) {
                    float pxaa = px + (xa / (float)SAMPLE_GRID_SIZE / resolution);
                    float pyaa = py + (ya / (float)SAMPLE_GRID_SIZE / resolution);
                    if(!insideLine(pxaa, pyaa, ax, ay, bx, by, lineThickness)) {
                        val += 1.0f;
                    }
                    max += 1.0f;
                }
            }
        }
        
        val /= max;
        d_weaveBlock[(z * resolution * resolution) + (y * resolution) + x] = d_currentImage[(y * resolution) + x] * val;
    }

    return;

}

__device__ int ipowi(int base, int power) {
    int num = 1;
    for (size_t i = 0; i < power; i++)
    {
        num *= base;
    }
    return num;
}

__global__ void dev_calculateLoss(float* d_weaveBlock, 
    float* d_tempWeaveBlock,
    int* d_connectionMatrix,
    float* d_currentImage,
    float* d_targetImage,
    Point* d_points,
    float* d_scores,
    float* d_gausianKernel,
    int kernelSize,
    int currentPoint,
    int pointCount,
    int resolution) {

    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int z = (blockIdx.z * blockDim.z) + threadIdx.z;

    // Ensure thread is within bound
    if(z < pointCount && x < resolution && y < resolution) {

        // // Blur image
        float accum = 0.0f;
        for (int blurX = -kernelSize / 2; blurX <= kernelSize / 2; blurX++)
        {
            for (int blurY = -kernelSize / 2; blurY <= kernelSize / 2; blurY++)
            {
                float strength = d_gausianKernel[((blurY + kernelSize / 2) * kernelSize) + blurX + (kernelSize / 2)];
                int xi = x + blurX;
                int yi = y + blurY;
                if(xi < 0 || xi >= resolution || yi < 0 || yi >= resolution) {
                    accum += strength;
                } else {
                    accum += strength * d_weaveBlock[(z * resolution * resolution) + (yi * resolution) + xi];
                }
            }
        }
        // float accum = d_weaveBlock[(z * resolution * resolution) + (y * resolution) + x];

        // Get Pixel loss
        float l1 = accum - d_targetImage[(y * resolution) + x];
        float l2 = l1 * l1;
        // if(z == 45)
        //     d_currentImage[(y * resolution) + x] = l2;

        float loss = 0.0f;
        loss += l2;

        // If the connection already exists, penalize
        if(d_connectionMatrix[(z * resolution) + currentPoint] == 1){
            loss += 1.0f;
        }

        // If connection is close, penalize proportionally
        int apart = 1 + fminf(labs(z - currentPoint), labs(z - 100 - currentPoint));
        float closenessPenalty = 1.0f / expf(apart);
        loss += closenessPenalty;

        int idx = (threadIdx.y * blockDim.x) + threadIdx.x;
        __shared__ float blockLoss[32 * 32];
        blockLoss[idx] = loss;

        // Reduce sum
        for (size_t s = 1; s < (blockDim.x * blockDim.y); s *= 2)
        {
            __syncthreads();
            if(idx % (2*s) == 0) {
                blockLoss[idx] += blockLoss[idx + s];
            }
        }
        
        __syncthreads();

        if(threadIdx.x == 0 && threadIdx.y == 0){
            atomicAdd(&d_scores[z], blockLoss[0]);
            // float l = 0.0f;
            // for (size_t a = 0; a < blockDim.x * blockDim.y; a++)
            // {
            //     l+= blockLoss[a];
            // }
            // atomicAdd(&d_scores[z], l);
        }
            // atomicAdd(&d_scores[z], loss);

        // 115810 - 115757 -
    }

    // if(z == 49) {
    //     d_currentImage[(y * resolution) + x] = d_tempWeaveBlock[(z * resolution * resolution) + (y * resolution) + x];
    // }
}